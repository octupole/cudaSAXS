#include <iostream>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>

extern "C" {
#define CHECK_CUDA_ERROR(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA Error: " << hipGetErrorString(err) << " at line " << __LINE__ << std::endl; \
            exit(err); \
        } \
    } while (0)

#define CHECK_CUFFT_ERROR(call) \
    do { \
        hipfftResult err = call; \
        if (err != HIPFFT_SUCCESS) { \
            std::cerr << "CUFFT Error: " << err << " at line " << __LINE__ << std::endl; \
            exit(err); \
        } \
    } while (0)

void fft3d(float* h_data, int nx, int ny, int nz, bool inverse) {
    hipfftHandle plan;
    hipfftComplex* d_data;
    size_t size = nx * ny * nz * sizeof(float);
    size_t complex_size = nx * ny * (nz / 2 + 1) * sizeof(hipfftComplex);

    // Allocate device memory
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_data, complex_size));
    if (inverse) {
        // Copy data to device
        CHECK_CUDA_ERROR(hipMemcpy(d_data, h_data, complex_size, hipMemcpyHostToDevice));

        // Create a 3-D FFT plan for inverse transform
        CHECK_CUFFT_ERROR(hipfftPlan3d(&plan, nx, ny, nz, HIPFFT_C2R));

        // Execute the inverse FFT
        CHECK_CUFFT_ERROR(hipfftExecC2R(plan, d_data, (hipfftReal*)d_data));

        // Copy the result back to host
        CHECK_CUDA_ERROR(hipMemcpy(h_data, d_data, size, hipMemcpyDeviceToHost));

        // Normalize the result
        for (int i = 0; i < nx * ny * nz; ++i) {
            h_data[i] /= (nx * ny * nz);
        }
    } else {
        // Copy data to device
        CHECK_CUDA_ERROR(hipMemcpy(d_data, h_data, size, hipMemcpyHostToDevice));
        // Create a 3-D FFT plan for forward transform
        CHECK_CUFFT_ERROR(hipfftPlan3d(&plan, nx, ny, nz, HIPFFT_R2C));

        // Execute the forward FFT
        CHECK_CUFFT_ERROR(hipfftExecR2C(plan, (hipfftReal*)d_data, d_data));


        // Copy the result back to host
        CHECK_CUDA_ERROR(hipMemcpy(h_data, d_data, complex_size, hipMemcpyDeviceToHost));
        }

    // Clean up
    hipfftDestroy(plan);
    hipFree(d_data);
}
}