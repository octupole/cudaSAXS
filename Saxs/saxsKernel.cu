#include "hip/hip_runtime.h"
#include "saxsKernel.h"
#include "BSpmod.h"
#include "Scattering.h"
#include "opsfact.h"
#include <hip/hip_runtime.h> // Include CUDA runtime header
/**
 * @brief Applies a modulus calculation to a grid of complex values.
 *
 * This kernel function calculates the modulus of each complex value in the input grid
 * and stores the result in the output grid.
 *
 * @param grid_q The input grid of complex values.
 * @param modX The modulus values for the x-dimension.
 * @param modY The modulus values for the y-dimension.
 * @param modZ The modulus values for the z-dimension.
 * @param numParticles The number of particles.
 * @param nnx The number of grid points in the x-dimension.
 * @param nny The number of grid points in the y-dimension.
 * @param nnz The number of grid points in the z-dimension.
 */
__global__ void modulusKernel(hipFloatComplex *grid_q, float *modX, float *modY, float *modZ,
                              int numParticles, int nnx, int nny, int nnz)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;
    int nx0 = static_cast<int>(nnx);
    int ny0 = static_cast<int>(nny);
    int nz0 = static_cast<int>(nnz);
    if (i < nx0 && j < ny0 && k < (nz0 / 2 + 1))
    {
        int idx = i + j * nx0 + k * nx0 * ny0;
        float bsp_i = modX[i];
        float bsp_j = modX[j];
        float bsp_k = modX[k];

        hipFloatComplex bsp = make_hipComplex(bsp_i * bsp_j * bsp_k, 0.0f);
        hipFloatComplex conj = hipConjf(grid_q[idx]);
        hipFloatComplex product = hipCmulf(conj, bsp);
        hipFloatComplex D = make_hipComplex(1.0f / (float)numParticles, 0.0f);
        grid_q[idx] = hipCmulf(hipCmulf(grid_q[idx], product), D);
    }
}
/**
 * @brief Performs scattering calculations on a grid of complex values.
 *
 * This kernel function calculates the scattering contribution for each grid point
 * based on the provided scattering factors and the grid of complex values.
 *
 * @param grid_q The input grid of complex values.
 * @param grid_oq The output grid of complex values.
 * @param oc The orientation coefficients.
 * @param Scatter The scattering factors.
 * @param nnx The number of grid points in the x-dimension.
 * @param nny The number of grid points in the y-dimension.
 * @param nnz The number of grid points in the z-dimension.
 */
__global__ void scatterKernel(hipFloatComplex *grid_q, hipFloatComplex *grid_oq, float *oc,
                              /**
                               * @brief Performs scattering calculations on a grid of complex values.
                               *
                               * This kernel function calculates the scattering contribution for each grid point
                               * based on the provided scattering factors and the grid of complex values.
                               *
                               * @param grid_q The input grid of complex values.
                               * @param grid_oq The output grid of complex values.
                               * @param oc The orientation coefficients.
                               * @param Scatter The scattering factors.
                               * @param nnx The number of grid points in the x-dimension.
                               * @param nny The number of grid points in the y-dimension.
                               * @param nnz The number of grid points in the z-dimension.
                               */
                              float *Scatter, int nnx, int nny, int nnz)
{

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;
    int nx0 = static_cast<int>(nnx);
    int ny0 = static_cast<int>(nny);
    int nz0 = static_cast<int>(nnz);
    int nfx = (nx0 % 2 == 0) ? nx0 / 2 : nx0 / 2 + 1;
    int nfy = (ny0 % 2 == 0) ? ny0 / 2 : ny0 / 2 + 1;
    int nfz = (nz0 % 2 == 0) ? nz0 / 2 : nz0 / 2 + 1;
    if (i < nx0 && j < ny0 && k < (nz0 / 2 + 1))
    {
        int idx = i + j * nx0 + k * nx0 * ny0;
        // printf("i: %d, j: %d, k: %d\n", i, j, k);
        opsfact ff;
        ff.allocate_device(Scatter);
        int ia = (i < nfx) ? i : i - nx0;
        int ja = (j < nfy) ? j : j - ny0;
        int ka = (k < nfz) ? k : k - nz0;
        float mw1, mw2, mw3, mw;
        mw1 = oc[XX * DIM + XX] * ia + oc[XX * DIM + YY] * ja + oc[XX * DIM + ZZ] * ka;
        mw2 = oc[YY * DIM + XX] * ia + oc[YY * DIM + YY] * ja + oc[YY * DIM + ZZ] * ka;
        mw3 = oc[ZZ * DIM + XX] * ia + oc[ZZ * DIM + YY] * ja + oc[ZZ * DIM + ZZ] * ka;
        mw1 = 2.0 * M_PI * mw1;
        mw2 = 2.0 * M_PI * mw2;
        mw3 = 2.0 * M_PI * mw3;
        mw = sqrt(mw1 * mw1 + mw2 * mw2 + mw3 * mw3);
        hipFloatComplex fq = make_hipComplex(ff(mw), 0.0f);

        grid_oq[idx] = hipCaddf(grid_oq[idx], hipCmulf(fq, grid_q[idx]));
    }
}
/**
 * @brief Computes the density contribution of each particle to the grid.
 *
 * This kernel function calculates the density contribution of each particle to the grid
 * using B-spline interpolation. It iterates over the grid points within the support
 * of the particle and adds the contribution to the corresponding grid points.
 *
 * @param xa The array of particle coordinates.
 * @param grid The grid to store the density contributions.
 * @param order The order of the B-spline interpolation.
 * @param numParticles The number of particles.
 * @param nx The number of grid points in the x-dimension.
 * @param ny The number of grid points in the y-dimension.
 * @param nz The number of grid points in the z-dimension.
 */
__global__ void rhoKernel(float *xa, float *grid, int order, int numParticles, int nx, int ny, int nz)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < numParticles)
    {

        float g_x[DIM][MAX_ORDER], g_dx[DIM][MAX_ORDER];
        Splines bsplineX;
        Splines bsplineY;
        Splines bsplineZ;
        bsplineX.allocate_device(&g_x[XX][0], &g_dx[XX][0]);
        bsplineY.allocate_device(&g_x[YY][0], &g_dx[YY][0]);
        bsplineZ.allocate_device(&g_x[ZZ][0], &g_dx[ZZ][0]);

        int nx0 = static_cast<int>(nx);
        int ny0 = static_cast<int>(ny);
        int nz0 = static_cast<int>(nz);
        float x1, y1, z1, r1, s1, t1, gx, gy, gz;
        int mx, my, mz;

        x1 = xa[idx * DIM + XX];
        y1 = xa[idx * DIM + YY];
        z1 = xa[idx * DIM + ZZ];
        r1 = static_cast<float>(nx0 * (x1 - rint(x1 - 0.5)));
        s1 = static_cast<float>(ny0 * (y1 - rint(y1 - 0.5)));
        t1 = static_cast<float>(nz0 * (z1 - rint(z1 - 0.5)));
        mx = static_cast<int>(r1);
        my = static_cast<int>(s1);
        mz = static_cast<int>(t1);

        gx = r1 - static_cast<float>(mx);
        gy = s1 - static_cast<float>(my);
        gz = t1 - static_cast<float>(mz);
        spline splX = bsplineX(gx);
        spline splY = bsplineX(gy);
        spline splZ = bsplineX(gz);

        int i0 = mx - order;

        for (auto o = 0; o < order; o++)
        {
            int i = i0 + (nx0 - ((i0 >= 0) ? nx0 : -nx0)) / 2;

            int j0 = my - order;
            for (auto p = 0; p < order; p++)
            {
                int j = j0 + (ny0 - ((j0 >= 0) ? ny0 : -ny0)) / 2;

                int k0 = mz - order;
                for (auto q = 0; q < order; q++)
                {
                    int k = k0 + (nz0 - ((k0 >= 0) ? nz0 : -nz0)) / 2;
                    float fact_o = splX.x[o];
                    float fact_p = fact_o * splY.x[p];
                    float fact_q = fact_p * splZ.x[q];
                    int ig = i + j * nx0 + k * nx0 * ny0;
                    atomicAdd(&grid[ig], fact_q);
                    k0++;
                }
                j0++;
            }
            i0++;
        }
    }
}
/**
 * @brief Kernel function to initialize a 3D grid with a given density value.
 *
 * This kernel function is used to initialize a 3D grid with a given density value. The grid is represented as a 1D array, and the kernel function calculates the 1D index from the 3D coordinates of each grid point.
 *
 * @param d_grid Pointer to the 1D array representing the 3D grid.
 * @param myDens The density value to be assigned to the grid.
 * @param nx The size of the grid in the x-dimension.
 * @param ny The size of the grid in the y-dimension.
 * @param nz The size of the grid in the z-dimension.
 * @param nnx The size of the super-sampled grid in the x-dimension.
 * @param nny The size of the super-sampled grid in the y-dimension.
 * @param nnz The size of the super-sampled grid in the z-dimension.
 */
__global__ void superDensityKernel(float *d_grid, float *d_gridSup, float myDens, int nx, int ny, int nz, int nnx, int nny, int nnz)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    int nx0 = static_cast<int>(nnx);
    int ny0 = static_cast<int>(nny);
    int nz0 = static_cast<int>(nnz);
    if (x < nx0 && y < ny0 && z < nz0)
    {
        int idx_s = x + y * nx0 + z * nx0 * ny0;
        d_gridSup[idx_s] = myDens;
        if (x < nx && y < ny && z < nz)
        {
            int idx = x + y * nx + z * nx * ny;
            d_gridSup[idx_s] = d_grid[idx];
        }
    }
}
/**
 * @brief Initializes a 3D grid of complex numbers with zero values.
 *
 * This CUDA kernel function initializes a 3D grid of hipFloatComplex values to zero. The grid is represented as a 1D array, and the kernel function calculates the 1D index from the 3D coordinates of each grid point.
 *
 * @param d_grid Pointer to the 1D array representing the 3D grid of hipFloatComplex values.
 * @param nx The size of the grid in the x-dimension.
 * @param ny The size of the grid in the y-dimension.
 * @param nz The size of the grid in the z-dimension.
 */
__global__ void initializeIqKernel(hipFloatComplex *d_grid, int nx, int ny, int nz)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    int nx0 = static_cast<int>(nx);
    int ny0 = static_cast<int>(ny);
    int nz0 = static_cast<int>(nz);
    if (x < nx0 && y < ny0 && z < nz0)
    {
        int idx = x + y * nx0 + z * nx0 * ny0;
        d_grid[idx] = make_hipComplex(0.0f, 0.0f);
    }
}
/**
 * @brief Initializes a 3D grid of floating-point values to zero.
 *
 * This CUDA kernel function initializes a 3D grid of floating-point values to zero. The grid is represented as a 1D array, and the kernel function calculates the 1D index from the 3D coordinates of each grid point.
 *
 * @param d_grid Pointer to the 1D array representing the 3D grid of floating-point values.
 * @param nx The size of the grid in the x-dimension.
 * @param ny The size of the grid in the y-dimension.
 * @param nz The size of the grid in the z-dimension.
 */
__global__ void initializeDensityKernel(float *d_grid, int nx, int ny, int nz)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    int nx0 = static_cast<int>(nx);
    int ny0 = static_cast<int>(ny);
    int nz0 = static_cast<int>(nz);
    if (x < nx0 && y < ny0 && z < nz0)
    {
        int idx = x + y * nx0 + z * nx0 * ny0;
        d_grid[idx] = 0.0f;
    }
}
/**
 * @brief Performs padding on a 3D grid, computing the average density and count of points on the border.
 *
 * This CUDA kernel function performs padding on a 3D grid, computing the average density and count of points on the border of the grid. The grid is represented as a 1D array, and the kernel function calculates the 1D index from the 3D coordinates of each grid point.
 *
 * @param grid Pointer to the 1D array representing the 3D grid of floating-point values.
 * @param nx The size of the grid in the x-dimension.
 * @param ny The size of the grid in the y-dimension.
 * @param nz The size of the grid in the z-dimension.
 * @param dx The padding size in the x-dimension.
 * @param dy The padding size in the y-dimension.
 * @param dz The padding size in the z-dimension.
 * @param Dens Pointer to a device-side float variable to store the total density of the border points.
 * @param count Pointer to a device-side integer variable to store the count of border points.
 */
__global__ void paddingKernel(float *grid, int nx, int ny, int nz, int dx, int dy, int dz, float *Dens, int *count)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    int nx0 = static_cast<int>(nx);
    int ny0 = static_cast<int>(ny);
    int nz0 = static_cast<int>(nz);
    int mx = nx - dx;
    int my = ny - dy;
    int mz = nz - dz;
    if (x < nx0 && y < ny0 && z < nz0)
    {
        int idx = x + y * nx0 + z * nx0 * ny0;
        bool cond1 = (x > dx && x < mx) && (y > dy && y < my) && (z > dz && z < mz);
        if (!cond1)
        {
            atomicAdd(count, 1);
            atomicAdd(Dens, grid[idx]);
        }
    }
}

/**
 * Processes a set of particles and computes their contribution to the SAXS intensity.
 *
 * This function iterates over a set of particles, transforms their coordinates based on the orientation matrix,
 * and computes their contribution to the SAXS intensity. It then performs padding, supersampling, and Fourier
 * transform operations on the density grid to compute the final SAXS intensity.
 *
 * @param coords A vector of particle coordinates.
 * @param index_map A map of particle indices, where the keys are particle types and the values are vectors of indices.
 * @param oc The orientation matrix.
 */
void saxsKernel::runPKernel(std::vector<std::vector<float>> &coords, std::map<std::string, std::vector<int>> &index_map, std::vector<std::vector<float>> &oc)
{
    hipfftHandle plan;
    hipfftPlan3d(&plan, nnx, nny, nnz, HIPFFT_R2C);
    // Cudaevents

    // hipEvent_t start, stop;
    // hipEventCreate(&start);
    // hipEventCreate(&stop);
    // hipEventRecord(start);

    // to compute average density on the border
    thrust::host_vector<float> h_Dens = {0.0f};
    thrust::host_vector<int> h_count = {0};
    thrust::device_vector<float> d_Dens = h_Dens;
    thrust::device_vector<int> d_count = h_count;
    int mx = borderBins(nx, SHELL);
    int my = borderBins(ny, SHELL);
    int mz = borderBins(nz, SHELL);
    thrust::host_vector<float> h_oc(DIM * DIM);
    for (int i = 0; i < DIM; ++i)
        for (int j = 0; j < DIM; ++j)
            h_oc[i * DIM + j] = oc[i][j];

    thrust::device_vector<float> d_oc = h_oc;
    float *d_oc_ptr = thrust::raw_pointer_cast(d_oc.data());

    dim3 blockDim(npx, npy, npz);
    dim3 gridDim((nnx + blockDim.x - 1) / blockDim.x,
                 (nny + blockDim.y - 1) / blockDim.y,
                 (nnz + blockDim.z - 1) / blockDim.z);
    auto nnpz = nnz / 2 + 1;
    initializeIqKernel<<<gridDim, blockDim>>>(d_gridSupC_ptr, nnx, nny, nnpz);

    /**
     * Processes a set of particles and computes their contribution to the SAXS intensity.
     *
     * This function iterates over a set of particles, transforms their coordinates based on the orientation matrix,
     * and computes their contribution to the SAXS intensity. It then performs padding, supersampling, and Fourier
     * transform operations on the density grid to compute the final SAXS intensity.
     *
     * @param coords A vector of particle coordinates.
     * @param index_map A map of particle indices, where the keys are particle types and the values are vectors of indices.
     * @param oc The orientation matrix.
     */
    for (const auto &pair : index_map)
    {
        std::string type = pair.first;
        std::vector<int> value = pair.second;
        std::vector<std::vector<float>> Particles;

        std::transform(value.begin(), value.end(), std::back_inserter(Particles), [&coords](int i)
                       { return coords[i]; });

        this->numParticles = Particles.size();

        // Allocate and copy particles to the device
        thrust::host_vector<float> h_particles(numParticles * 3);
        for (int i = 0; i < numParticles; ++i)
        {
            h_particles[i * 3] = oc[XX][XX] * Particles[i][XX] + oc[XX][YY] * Particles[i][YY] + oc[XX][ZZ] * Particles[i][ZZ];
            h_particles[i * 3 + 1] = oc[YY][XX] * Particles[i][XX] + oc[YY][YY] * Particles[i][YY] + oc[YY][ZZ] * Particles[i][ZZ];
            h_particles[i * 3 + 2] = oc[ZZ][XX] * Particles[i][XX] + oc[ZZ][YY] * Particles[i][YY] + oc[ZZ][ZZ] * Particles[i][ZZ];
        }

        thrust::device_vector<float> d_particles = h_particles;
        thrust::host_vector<float> h_scatter = Scattering::getScattering(type);
        thrust::device_vector<float> d_scatter = h_scatter;

        float *d_particles_ptr = thrust::raw_pointer_cast(d_particles.data());
        float *d_scatter_ptr = thrust::raw_pointer_cast(d_scatter.data());

        const int THREADS_PER_BLOCK = 256;
        int numBlocks = (numParticles + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

        initializeDensityKernel<<<gridDim, blockDim>>>(d_grid_ptr, nx, ny, nz);

        // Synchronize the device
        hipDeviceSynchronize();

        rhoKernel<<<numBlocks, THREADS_PER_BLOCK>>>(d_particles_ptr, d_grid_ptr, order,
                                                    numParticles, nx, ny, nz);
        // Synchronize the device
        hipDeviceSynchronize();

        paddingKernel<<<gridDim, blockDim>>>(d_grid_ptr, nx, ny, nz, mx, my, mz,
                                             thrust::raw_pointer_cast(d_Dens.data()),
                                             thrust::raw_pointer_cast(d_count.data()));
        // Synchronize the device
        hipDeviceSynchronize();

        h_Dens = d_Dens;
        h_count = d_count;
        float myDens = h_Dens[0] / (float)h_count[0];
        superDensityKernel<<<gridDim, blockDim>>>(d_grid_ptr, d_gridSup_ptr, myDens, nx, ny, nz, nnx, nny, nnz);

        // Synchronize the device
        hipDeviceSynchronize();

        hipfftExecR2C(plan, d_gridSup_ptr, (hipFloatComplex *)d_gridSup_ptr);

        // Synchronize the device
        hipDeviceSynchronize();

        scatterKernel<<<gridDim, blockDim>>>((hipFloatComplex *)d_gridSup_ptr, d_gridSupC_ptr, d_oc_ptr, d_scatter_ptr, nnx, nny, nnz);

        // Synchronize the device
        hipDeviceSynchronize();
        modulusKernel<<<gridDim, blockDim>>>(d_gridSupC_ptr, d_moduleX_ptr, d_moduleY_ptr, d_moduleZ_ptr, numParticles, nnx, nny, nnz);

        // Synchronize the device
        hipDeviceSynchronize();
    }
    // hipDeviceSynchronize();
    // hipEventRecord(stop);
    // hipEventSynchronize(stop);

    // // Calculate the elapsed time in milliseconds
    // float gpuElapsedTime;
    // hipEventElapsedTime(&gpuElapsedTime, start, stop);

    // // Destroy the events
    // hipEventDestroy(start);
    // hipEventDestroy(stop);
}

/**
 * @brief Creates the necessary memory for the SAXS computation.
 *
 * This function sets up the memory buffers and allocates memory for the SAXS computation.
 * It calculates the optimal grid sizes (nnx, nny, nnz) based on the original grid sizes (nx, ny, nz)
 * and the given sigma value. It then creates the necessary host and device memory buffers for the
 * grid, super-grid, and module data.
 *
 * @param[in,out] nnx The optimal x-dimension of the super-grid.
 * @param[in,out] nny The optimal y-dimension of the super-grid.
 * @param[in,out] nnz The optimal z-dimension of the super-grid.
 * @param[in] sigma The sigma value used to calculate the optimal grid sizes.
 */
void saxsKernel::createMemory(int &nnx, int &nny, int &nnz, float sigma)
{
    this->sigma = sigma;
    if (nnx == 0)
    {
        nnx = this->nnx = static_cast<int>(findClosestProduct(nx, sigma));
        nny = this->nny = static_cast<int>(findClosestProduct(ny, sigma));
        nnz = this->nnz = static_cast<int>(findClosestProduct(nz, sigma));
    }
    else
    {
        this->nnx = nnx;
        this->nny = nny;
        this->nnz = nnz;
    }

    size_t nnpz = nnz / 2 + 1;

    BSpline::BSpmod *bsp_modx = new BSpline::BSpmod(nx, ny, nz);
    std::cout << "Cell with nx: " << nx << " ny: " << ny << " nz: " << nz << std::endl;
    std::cout << "SuperCell with nnx: " << nnx << " nny: " << nny << " nnz: " << nnz << std::endl;

    thrust::host_vector<float> h_moduleX = bsp_modx->ModX();
    thrust::host_vector<float> h_moduleY = bsp_modx->ModY();
    thrust::host_vector<float> h_moduleZ = bsp_modx->ModZ();

    d_moduleX = h_moduleX;
    d_moduleY = h_moduleY;
    d_moduleZ = h_moduleZ;
    d_moduleX_ptr = thrust::raw_pointer_cast(d_moduleX.data());
    d_moduleY_ptr = thrust::raw_pointer_cast(d_moduleY.data());
    d_moduleZ_ptr = thrust::raw_pointer_cast(d_moduleZ.data());

    thrust::host_vector<float> h_grid(nx * ny * nz);
    thrust::host_vector<float> h_gridSup(2 * nnx * nny * nnpz);
    thrust::host_vector<hipFloatComplex> h_gridSupC(nnx * nny * nnpz);

    d_grid = h_grid;
    d_gridSup = h_gridSup;
    d_gridSupC = h_gridSupC;

    d_grid_ptr = thrust::raw_pointer_cast(d_grid.data());
    d_gridSup_ptr = thrust::raw_pointer_cast(d_gridSup.data());
    d_gridSupC_ptr = thrust::raw_pointer_cast(d_gridSupC.data());
    // Do bspmod
}
/**
 * Generates a vector of multiples of 2, 3, 5, and 7 up to a given limit.
 *
 * This function generates all possible multiples of 2, 3, 5, and 7 up to the
 * specified limit, and returns them as a sorted, unique vector.
 *
 * @param limit The maximum value to generate multiples up to.
 * @return A vector of all multiples of 2, 3, 5, and 7 up to the given limit.
 */
// Function to generate multiples of 2, 3, 5, and 7 up to a given limit
std::vector<long long> saxsKernel::generateMultiples(long long limit)
{
    std::vector<long long> multiples;
    for (int a = 0; std::pow(2, a) <= limit; ++a)
    {
        for (int b = 0; std::pow(2, a) * std::pow(3, b) <= limit; ++b)
        {
            for (int c = 0; std::pow(2, a) * std::pow(3, b) * std::pow(5, c) <= limit; ++c)
            {
                for (int d = 0; std::pow(2, a) * std::pow(3, b) * std::pow(5, c) * std::pow(7, d) <= limit; ++d)
                {
                    long long multiple = std::pow(2, a) * std::pow(3, b) * std::pow(5, c) * std::pow(7, d);
                    if (multiple <= limit)
                    {
                        multiples.push_back(multiple);
                    }
                }
            }
        }
    }
    std::sort(multiples.begin(), multiples.end());
    multiples.erase(std::unique(multiples.begin(), multiples.end()), multiples.end());
    return multiples;
}

/**
 * Finds the closest integer to N * sigma that is obtainable by multiplying only 2, 3, 5, and 7.
 *
 * This function takes a target value N and a standard deviation sigma, and finds the closest integer
 * to N * sigma that can be expressed as a product of only the prime factors 2, 3, 5, and 7.
 *
 * @param n The target value N.
 * @param sigma The standard deviation.
 * @return The closest integer to N * sigma that is obtainable by multiplying only 2, 3, 5, and 7.
 */
// Function to find the closest integer to N * sigma that is obtainable by multiplying only 2, 3, 5, and 7
long long saxsKernel::findClosestProduct(int n, double sigma)
{
    long long target = std::round(n * sigma);
    long long limit = target * 2; // A generous limit for generating multiples
    std::vector<long long> multiples = generateMultiples(limit);

    long long closest = target;
    long long minDifference = std::numeric_limits<long long>::max();

    for (long long multiple : multiples)
    {
        long long difference = std::abs(multiple - target);
        if (difference < minDifference)
        {
            minDifference = difference;
            closest = multiple;
        }
    }

    return closest;
}

saxsKernel::~saxsKernel()
{
}
