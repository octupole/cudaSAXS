#include "hip/hip_runtime.h"
#include "saxsKernel.h"
#include "BSpmod.h"
#include "Scattering.h"
#include "opsfact.h"
#include <hip/hip_runtime.h> // Include CUDA runtime header
#include <hip/hip_complex.h>
// Kernel to calculate |K| values and populate the histogram
__global__ void calculate_histogram(hipFloatComplex *d_array, float *d_histogram, float *d_nhist, float *oc, int nx, int ny, int nz,
                                    float bin_size, float qcut, int num_bins)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;
    int npz = nz / 2 + 1;
    if (i < nx && j < ny && k < npz)
    { // Only consider the upper half in z-direction

        int nfx = (nx % 2 == 0) ? nx / 2 : nx / 2 + 1;
        int nfy = (ny % 2 == 0) ? ny / 2 : ny / 2 + 1;
        int nfz = (nz % 2 == 0) ? nz / 2 : nz / 2 + 1;

        int ia = (i < nfx) ? i : i - nx;
        int ja = (j < nfy) ? j : j - ny;
        int ka = (k < nfz) ? k : k - nz;
        int ib = i == 0 ? 0 : nx - i;
        int jb = j == 0 ? 0 : ny - j;
        float mw1, mw2, mw3, mw;
        mw1 = oc[XX * DIM + XX] * ia + oc[XX * DIM + YY] * ja + oc[XX * DIM + ZZ] * ka;
        mw1 = 2.0 * M_PI * mw1;
        mw2 = oc[YY * DIM + XX] * ia + oc[YY * DIM + YY] * ja + oc[YY * DIM + ZZ] * ka;
        mw2 = 2.0 * M_PI * mw2;
        mw3 = oc[ZZ * DIM + XX] * ia + oc[ZZ * DIM + YY] * ja + oc[ZZ * DIM + ZZ] * ka;
        mw3 = 2.0 * M_PI * mw3;
        mw = sqrtf(mw1 * mw1 + mw2 * mw2 + mw3 * mw3);
        if (mw > qcut)
            return;
        int h0 = static_cast<int>(mw / bin_size);
        int h1 = h0 + 1;
        hipFloatComplex v0;
        if (h0 < num_bins)
        {
            int idx = k + j * npz + i * npz * ny;
            int idbx = k + jb * npz + ib * npz * ny;
            v0 = d_array[idx];
            if (k != 0 && k != npz - 1)
            {
                auto v1 = d_array[idbx];
                v0 = hipCaddf(v0, v1);
                v0 = hipCmulf(v0, make_hipFloatComplex(0.5f, 0.0f));
            }
            atomicAdd(&d_histogram[h0], hipCrealf(v0));
            atomicAdd(&d_nhist[h0], 1.0f);
            if (h0 != 0)
            {
                atomicAdd(&d_histogram[h1], hipCrealf(v0));
                atomicAdd(&d_nhist[h1], 1.0f);
            }
        }
    }
}

/**
 * @brief Applies a modulus calculation to a grid of complex values.
 *
 * This kernel function calculates the modulus of each complex value in the input grid
 * and stores the result in the output grid.
 *
 * @param grid_q The input grid of complex values.
 * @param modX The modulus values for the x-dimension.
 * @param modY The modulus values for the y-dimension.
 * @param modZ The modulus values for the z-dimension.
 * @param numParticles The number of particles.
 * @param nnx The number of grid points in the x-dimension.
 * @param nny The number of grid points in the y-dimension.
 * @param nnz The number of grid points in the z-dimension.
 */
__global__ void modulusKernel(hipFloatComplex *grid_q, float *modX, float *modY, float *modZ,
                              int numParticles, int nnx, int nny, int nnz)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;
    int nnpz = nnz / 2 + 1;
    if (i < nnx && j < nny && k < nnpz)
    {
        int idx = k + j * nnpz + i * nnpz * nny;
        float bsp_i = modX[i];
        float bsp_j = modX[j];
        float bsp_k = modX[k];
        float bsp_ijk = bsp_i * bsp_j * bsp_k / (float)numParticles;
        hipFloatComplex bsp = make_hipComplex(bsp_ijk, 0.0f);
        grid_q[idx] = hipCmulf(hipConjf(grid_q[idx]), grid_q[idx]);
        grid_q[idx] = hipCmulf(grid_q[idx], bsp);
    }
}
/**
 * @brief Performs scattering calculations on a grid of complex values.
 *
 * This kernel function calculates the scattering contribution for each grid point
 * based on the provided scattering factors and the grid of complex values.
 *
 * @param grid_q The input grid of complex values.
 * @param grid_oq The output grid of complex values.
 * @param oc The orientation coefficients.
 * @param Scatter The scattering factors.
 * @param nnx The number of grid points in the x-dimension.
 * @param nny The number of grid points in the y-dimension.
 * @param nnz The number of grid points in the z-dimension.
 */
__global__ void scatterKernel(hipFloatComplex *grid_q, hipFloatComplex *grid_oq, float *oc,
                              float *Scatter, int nnx, int nny, int nnz, float qcut)
{

    // if (idx >= nx0 * ny0 * (nz0 / 2 + 1))
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;
    int nfx = (nnx % 2 == 0) ? nnx / 2 : nnx / 2 + 1;
    int nfy = (nny % 2 == 0) ? nny / 2 : nny / 2 + 1;
    int nfz = (nnz % 2 == 0) ? nnz / 2 : nnz / 2 + 1;
    int nnpz = nnz / 2 + 1;
    if (i < nnx && j < nny && k < nnpz)
    {
        int idx = k + j * nnpz + i * nnpz * nny;

        opsfact ff;
        ff.allocate_device(Scatter);
        int ia = (i < nfx) ? i : i - nnx;
        int ja = (j < nfy) ? j : j - nny;
        int ka = (k < nfz) ? k : k - nnz;
        float mw1, mw2, mw3, mw;
        mw1 = oc[XX * DIM + XX] * ia + oc[XX * DIM + YY] * ja + oc[XX * DIM + ZZ] * ka;
        mw2 = oc[YY * DIM + XX] * ia + oc[YY * DIM + YY] * ja + oc[YY * DIM + ZZ] * ka;
        mw3 = oc[ZZ * DIM + XX] * ia + oc[ZZ * DIM + YY] * ja + oc[ZZ * DIM + ZZ] * ka;
        mw1 = 2.0 * M_PI * mw1;
        mw2 = 2.0 * M_PI * mw2;
        mw3 = 2.0 * M_PI * mw3;
        mw = sqrt(mw1 * mw1 + mw2 * mw2 + mw3 * mw3);
        if (mw > qcut)
            return;
        hipFloatComplex fq = make_hipComplex(ff(mw), 0.0f);
        hipFloatComplex mult = hipCmulf(fq, grid_q[idx]);
        grid_oq[idx] = hipCaddf(grid_oq[idx], mult);
    }
}
__global__ void zeroDensityKernel(float *d_grid, size_t size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < (int)size)
    {
        d_grid[idx] = 0.0f;
    }
}
__global__ void zeroDensityKernel(hipFloatComplex *d_grid, size_t size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < (int)size)
    {
        d_grid[idx] = make_hipComplex(0.0f, 0.0f);
    }
}

/**
 * @brief Computes the density contribution of each particle to the grid.
 *
 * This kernel function calculates the density contribution of each particle to the grid
 * using B-spline interpolation. It iterates over the grid points within the support
 * of the particle and adds the contribution to the corresponding grid points.
 *
 * @param xa The array of particle coordinates.
 * @param grid The grid to store the density contributions.
 * @param order The order of the B-spline interpolation.
 * @param numParticles The number of particles.
 * @param nx The number of grid points in the x-dimension.
 * @param ny The number of grid points in the y-dimension.
 * @param nz The number of grid points in the z-dimension.
 */
__global__ void rhoKernel(float *xa, float *grid, int order, int numParticles, int nx, int ny, int nz)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < numParticles)
    {
        Splines bsplineX;
        Splines bsplineY;
        Splines bsplineZ;

        int nx0 = static_cast<int>(nx);
        int ny0 = static_cast<int>(ny);
        int nz0 = static_cast<int>(nz);
        float x1, y1, z1, r1, s1, t1, gx, gy, gz;
        int mx, my, mz;

        x1 = xa[idx * DIM + XX];
        y1 = xa[idx * DIM + YY];
        z1 = xa[idx * DIM + ZZ];
        r1 = static_cast<float>(nx0 * (x1 - rint(x1 - 0.5)));
        s1 = static_cast<float>(ny0 * (y1 - rint(y1 - 0.5)));
        t1 = static_cast<float>(nz0 * (z1 - rint(z1 - 0.5)));
        mx = static_cast<int>(r1);
        my = static_cast<int>(s1);
        mz = static_cast<int>(t1);

        gx = r1 - static_cast<float>(mx);
        gy = s1 - static_cast<float>(my);
        gz = t1 - static_cast<float>(mz);
        spline splX = bsplineX(gx);
        spline splY = bsplineX(gy);
        spline splZ = bsplineX(gz);
        int i0 = mx - order;

        for (auto o = 0; o < order; o++)
        {
            int i = i0 + (nx0 - ((i0 >= 0) ? nx0 : -nx0)) / 2;

            int j0 = my - order;
            for (auto p = 0; p < order; p++)
            {
                int j = j0 + (ny0 - ((j0 >= 0) ? ny0 : -ny0)) / 2;

                int k0 = mz - order;
                for (auto q = 0; q < order; q++)
                {
                    int k = k0 + (nz0 - ((k0 >= 0) ? nz0 : -nz0)) / 2;
                    float fact_o = splX.x[o];
                    float fact_p = fact_o * splY.x[p];
                    float fact_q = fact_p * splZ.x[q];
                    int ig = k + j * nz0 + i * nz0 * ny0;
                    atomicAdd(&grid[ig], fact_q);
                    k0++;
                }
                j0++;
            }
            i0++;
        }
    }
}
/**
 * @brief Kernel function to initialize a 3D grid with a given density value.
 *
 * This kernel function is used to initialize a 3D grid with a given density value. The grid is represented as a 1D array, and the kernel function calculates the 1D index from the 3D coordinates of each grid point.
 *
 * @param d_grid Pointer to the 1D array representing the 3D grid.
 * @param myDens The density value to be assigned to the grid.
 * @param nx The size of the grid in the x-dimension.
 * @param ny The size of the grid in the y-dimension.
 * @param nz The size of the grid in the z-dimension.
 * @param nnx The size of the super-sampled grid in the x-dimension.
 * @param nny The size of the super-sampled grid in the y-dimension.
 * @param nnz The size of the super-sampled grid in the z-dimension.
 */
__global__ void superDensityKernel(float *d_grid, float *d_gridSup, float myDens, int nx, int ny, int nz, int nnx, int nny, int nnz)
{
    float N1 = (float)nnx / (float)nx;
    float N2 = (float)nny / (float)ny;
    float N3 = (float)nnz / (float)nz;

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    float summ0 = -myDens * (N1 * N2 * N3 - 1.0) / (N1 * N2 * N3);
    if (x < nnx && y < nny && z < nnz)
    {
        int idx_s = z + y * nnz + x * nnz * nny;
        d_gridSup[idx_s] = myDens;
        if (x < nx && y < ny && z < nz)
        {
            int idx = z + y * nz + x * nz * ny;
            d_gridSup[idx_s] = d_grid[idx];
        }
        d_gridSup[idx_s] += summ0;
    }
}

/**
 * @brief Performs padding on a 3D grid, computing the average density and count of points on the border.
 *
 * This CUDA kernel function performs padding on a 3D grid, computing the average density and count of points on the border of the grid. The grid is represented as a 1D array, and the kernel function calculates the 1D index from the 3D coordinates of each grid point.
 *
 * @param grid Pointer to the 1D array representing the 3D grid of floating-point values.
 * @param nx The size of the grid in the x-dimension.
 * @param ny The size of the grid in the y-dimension.
 * @param nz The size of the grid in the z-dimension.
 * @param dx The padding size in the x-dimension.
 * @param dy The padding size in the y-dimension.
 * @param dz The padding size in the z-dimension.
 * @param Dens Pointer to a device-side float variable to store the total density of the border points.
 * @param count Pointer to a device-side integer variable to store the count of border points.
 */
__global__ void paddingKernel(float *grid, int nx, int ny, int nz, int dx, int dy, int dz, float *Dens, int *count)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    int mx = nx - dx;
    int my = ny - dy;
    int mz = nz - dz;
    if (x < nx && y < ny && z < nz)
    {
        int idx = z + y * nz + x * nz * ny;
        bool cond1 = (x > dx && x < mx) && (y > dy && y < my) && (z > dz && z < mz);
        if (!cond1)
        {
            atomicAdd(&count[0], 1);
            atomicAdd(&Dens[0], grid[idx]);
        }
    }
}

/**
 * Processes a set of particles and computes their contribution to the SAXS intensity.
 *
 * This function iterates over a set of particles, transforms their coordinates based on the orientation matrix,
 * and computes their contribution to the SAXS intensity. It then performs padding, supersampling, and Fourier
 * transform operations on the density grid to compute the final SAXS intensity.
 *
 * @param coords A vector of particle coordinates.
 * @param index_map A map of particle indices, where the keys are particle types and the values are vectors of indices.
 * @param oc The orientation matrix.
 */
void saxsKernel::runPKernel(int frame, float Time, std::vector<std::vector<float>> &coords, std::map<std::string, std::vector<int>> &index_map, std::vector<std::vector<float>> &oc)
{
    static bool firstTime = true;
    // Cudaevents

    // to compute average density on the border
    if (firstTime)
    {
        this->resetHistogramParameters(oc);
        this->createMemory();
        this->writeBanner();
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    int mx = borderBins(nx, SHELL);
    int my = borderBins(ny, SHELL);
    int mz = borderBins(nz, SHELL);
    float mySigma = (float)Options::nx / (float)Options::nnx;

    thrust::host_vector<float> h_oc(DIM * DIM);
    for (int i = 0; i < DIM; ++i)
        for (int j = 0; j < DIM; ++j)
        {
            h_oc[i * DIM + j] = mySigma * oc[i][j];
        }

    thrust::device_vector<float> d_oc = h_oc;
    float *d_oc_ptr = thrust::raw_pointer_cast(d_oc.data());

    dim3 blockDim(npx, npy, npz);
    dim3 gridDim((nnx + blockDim.x - 1) / blockDim.x,
                 (nny + blockDim.y - 1) / blockDim.y,
                 (nnz + blockDim.z - 1) / blockDim.z);
    dim3 gridDim0((nx + blockDim.x - 1) / blockDim.x,
                  (ny + blockDim.y - 1) / blockDim.y,
                  (nz + blockDim.z - 1) / blockDim.z);
    const int THREADS_PER_BLOCK = 256;
    int numBlocksGrid = (d_grid.size() + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    int numBlocksGridSuperC = (d_gridSupC.size() + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    int numBlocksGridSuperAcc = (d_gridSupAcc.size() + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    int numBlocksGridSuper = (d_gridSup.size() + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    // zeroes the Sup density grid
    zeroDensityKernel<<<numBlocksGridSuperAcc, THREADS_PER_BLOCK>>>(d_gridSupAcc_ptr, d_gridSupAcc.size());

    int totParticles = 0;
    std::string formatted_string = fmt::format("--> Frame: {:<7}  Time Step: {:.2f} fs", frame, Time);

    // Print the formatted string
    std::cout << formatted_string << std::endl;

    for (const auto &pair : index_map)
    {
        hipfftHandle plan;
        hipfftPlan3d(&plan, nnx, nny, nnz, HIPFFT_R2C);

        thrust::host_vector<float> h_Dens = {0.0f};
        thrust::host_vector<int> h_count = {0};
        thrust::device_vector<float> d_Dens = h_Dens;
        thrust::device_vector<int> d_count = h_count;
        std::string type = pair.first;
        std::vector<int> value = pair.second;
        std::vector<std::vector<float>> Particles;

        std::transform(value.begin(), value.end(), std::back_inserter(Particles), [&coords](int i)
                       { return coords[i]; });

        this->numParticles = Particles.size();
        totParticles += this->numParticles;
        // Allocate and copy particles to the device
        thrust::host_vector<float> h_particles(numParticles * 3);
        for (int i = 0; i < numParticles; ++i)
        {
            h_particles[i * 3] = oc[XX][XX] * Particles[i][XX] + oc[XX][YY] * Particles[i][YY] + oc[XX][ZZ] * Particles[i][ZZ];
            h_particles[i * 3 + 1] = oc[YY][XX] * Particles[i][XX] + oc[YY][YY] * Particles[i][YY] + oc[YY][ZZ] * Particles[i][ZZ];
            h_particles[i * 3 + 2] = oc[ZZ][XX] * Particles[i][XX] + oc[ZZ][YY] * Particles[i][YY] + oc[ZZ][ZZ] * Particles[i][ZZ];
        }

        thrust::device_vector<float> d_particles = h_particles;
        thrust::host_vector<float> h_scatter = Scattering::getScattering(type);
        thrust::device_vector<float> d_scatter = h_scatter;

        float *d_particles_ptr = thrust::raw_pointer_cast(d_particles.data());
        float *d_scatter_ptr = thrust::raw_pointer_cast(d_scatter.data());

        int numBlocks = (numParticles + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

        //    Kernels launch for the rhoKernel

        zeroDensityKernel<<<numBlocksGrid, THREADS_PER_BLOCK>>>(d_grid_ptr, d_grid.size());

        rhoKernel<<<numBlocks, THREADS_PER_BLOCK>>>(d_particles_ptr, d_grid_ptr, order,
                                                    numParticles, nx, ny, nz);

        // Synchronize the device
        hipDeviceSynchronize();
        paddingKernel<<<gridDim0, blockDim>>>(d_grid_ptr, nx, ny, nz, mx, my, mz,
                                              thrust::raw_pointer_cast(d_Dens.data()),
                                              thrust::raw_pointer_cast(d_count.data()));
        // Synchronize the device
        hipDeviceSynchronize();

        h_Dens = d_Dens;
        h_count = d_count;
        float myDens = h_Dens[0] / (float)h_count[0];
        // zeroes the Sup density grid
        zeroDensityKernel<<<numBlocksGridSuperC, THREADS_PER_BLOCK>>>(d_gridSupC_ptr, d_gridSupC.size());
        hipDeviceSynchronize();

        superDensityKernel<<<gridDim, blockDim>>>(d_grid_ptr, d_gridSup_ptr, myDens, nx, ny, nz, nnx, nny, nnz);

        // Synchronize the device
        hipDeviceSynchronize();

        hipfftExecR2C(plan, d_gridSup_ptr, d_gridSupC_ptr);
        hipDeviceSynchronize();

        // Synchronize the device
        scatterKernel<<<gridDim, blockDim>>>(d_gridSupC_ptr, d_gridSupAcc_ptr, d_oc_ptr, d_scatter_ptr, nnx, nny, nnz, kcut);
        hipDeviceSynchronize();
    }
    modulusKernel<<<gridDim, blockDim>>>(d_gridSupAcc_ptr, d_moduleX_ptr, d_moduleY_ptr, d_moduleZ_ptr, totParticles, nnx, nny, nnz);
    // // Synchronize the device
    hipDeviceSynchronize();
    calculate_histogram<<<gridDim, blockDim>>>(d_gridSupAcc_ptr, d_histogram_ptr, d_nhist_ptr, d_oc_ptr, nnx, nny, nnz,
                                               bin_size, kcut, num_bins);

    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate the elapsed time in milliseconds
    float gpuElapsedTime;
    hipEventElapsedTime(&gpuElapsedTime, start, stop);
    // std::cout << "GPU Elapsed Time: " << gpuElapsedTime << " ms" << std::endl;

    // Destroy the events
    hipEventDestroy(start);
    hipEventDestroy(stop);
    firstTime = false;
}
std::vector<std::vector<float>> saxsKernel::getSaxs()
{

    std::vector<std::vector<float>> saxs;
    thrust::host_vector<float> h_histogram = d_histogram;
    thrust::host_vector<float> h_nhist = d_nhist;
    for (auto o{0}; o < h_histogram.size(); o++)
    {
        if (h_nhist[o] != 0.0f)
        {
            vector<float> val = {o * this->bin_size, h_histogram[o] / h_nhist[o]};
            saxs.push_back(val);
        }
    }
    return saxs;
}

/**
 * @brief Creates the necessary memory for the SAXS computation.
 *
 * This function sets up the memory buffers and allocates memory for the SAXS computation.
 * It calculates the optimal grid sizes (nnx, nny, nnz) based on the original grid sizes (nx, ny, nz)
 * and the given sigma value. It then creates the necessary host and device memory buffers for the
 * grid, super-grid, and module data.
 *
 * @param[in,out] nnx The optimal x-dimension of the super-grid.
 * @param[in,out] nny The optimal y-dimension of the super-grid.
 * @param[in,out] nnz The optimal z-dimension of the super-grid.
 * @param[in] sigma The sigma value used to calculate the optimal grid sizes.
 */
void saxsKernel::createMemory()
{
    size_t nnpz = nnz / 2 + 1;

    this->bin_size = Options::Dq;
    this->kcut = Options::Qcut;

    this->num_bins = static_cast<int>(kcut / bin_size) + 1;
    thrust::host_vector<float> h_histogram(num_bins, 0.0f);
    thrust::host_vector<float> h_nhist(num_bins, 0.0f);

    d_histogram = h_histogram;
    d_nhist = h_nhist;
    d_histogram_ptr = thrust::raw_pointer_cast(d_histogram.data());
    d_nhist_ptr = thrust::raw_pointer_cast(d_nhist.data());
    BSpline::BSpmod *bsp_modx = new BSpline::BSpmod(nnx, nny, nnz);

    thrust::host_vector<float> h_moduleX = bsp_modx->ModX();
    thrust::host_vector<float> h_moduleY = bsp_modx->ModY();
    thrust::host_vector<float> h_moduleZ = bsp_modx->ModZ();

    d_moduleX = h_moduleX;
    d_moduleY = h_moduleY;
    d_moduleZ = h_moduleZ;
    d_moduleX_ptr = thrust::raw_pointer_cast(d_moduleX.data());
    d_moduleY_ptr = thrust::raw_pointer_cast(d_moduleY.data());
    d_moduleZ_ptr = thrust::raw_pointer_cast(d_moduleZ.data());

    thrust::host_vector<float> h_grid(nx * ny * nz);
    thrust::host_vector<float> h_gridSup(nnx * nny * nnz);
    thrust::host_vector<hipFloatComplex> h_gridSupC(nnx * nny * nnpz);
    thrust::host_vector<hipFloatComplex> h_gridSupAcc(nnx * nny * nnpz);

    d_grid = h_grid;
    d_gridSup = h_gridSup;
    d_gridSupC = h_gridSupC;
    d_gridSupAcc = h_gridSupAcc;

    d_grid_ptr = thrust::raw_pointer_cast(d_grid.data());
    d_gridSup_ptr = thrust::raw_pointer_cast(d_gridSup.data());
    d_gridSupC_ptr = thrust::raw_pointer_cast(d_gridSupC.data());
    d_gridSupAcc_ptr = thrust::raw_pointer_cast(d_gridSupAcc.data());
    // Do bspmod
}
/**
 * Generates a vector of multiples of 2, 3, 5, and 7 up to a given limit.
 *
 * This function generates all possible multiples of 2, 3, 5, and 7 up to the
 * specified limit, and returns them as a sorted, unique vector.
 *
 * @param limit The maximum value to generate multiples up to.
 * @return A vector of all multiples of 2, 3, 5, and 7 up to the given limit.
 */
// Function to generate multiples of 2, 3, 5, and 7 up to a given limit
std::vector<long long> saxsKernel::generateMultiples(long long limit)
{
    std::vector<long long> multiples;
    for (int a = 0; std::pow(2, a) <= limit; ++a)
    {
        for (int b = 0; std::pow(2, a) * std::pow(3, b) <= limit; ++b)
        {
            for (int c = 0; std::pow(2, a) * std::pow(3, b) * std::pow(5, c) <= limit; ++c)
            {
                for (int d = 0; std::pow(2, a) * std::pow(3, b) * std::pow(5, c) * std::pow(7, d) <= limit; ++d)
                {
                    long long multiple = std::pow(2, a) * std::pow(3, b) * std::pow(5, c) * std::pow(7, d);
                    if (multiple <= limit)
                    {
                        multiples.push_back(multiple);
                    }
                }
            }
        }
    }
    std::sort(multiples.begin(), multiples.end());
    multiples.erase(std::unique(multiples.begin(), multiples.end()), multiples.end());
    return multiples;
}

/**
 * Finds the closest integer to N * sigma that is obtainable by multiplying only 2, 3, 5, and 7.
 *
 * This function takes a target value N and a standard deviation sigma, and finds the closest integer
 * to N * sigma that can be expressed as a product of only the prime factors 2, 3, 5, and 7.
 *
 * @param n The target value N.
 * @param sigma The standard deviation.
 * @return The closest integer to N * sigma that is obtainable by multiplying only 2, 3, 5, and 7.
 */
// Function to find the closest integer to N * sigma that is obtainable by multiplying only 2, 3, 5, and 7
long long saxsKernel::findClosestProduct(int n, float sigma)
{
    long long target = std::round(n * sigma);
    long long limit = target * 2; // A generous limit for generating multiples
    std::vector<long long> multiples = generateMultiples(limit);

    long long closest = target;
    long long minDifference = std::numeric_limits<long long>::max();

    for (long long multiple : multiples)
    {
        long long difference = std::abs(multiple - target);
        if (difference < minDifference)
        {
            minDifference = difference;
            closest = multiple;
        }
    }

    return closest;
}
void saxsKernel::scaledCell()
{
    sigma = Options::sigma;
    if (Options::nnx == 0)
    {
        nnx = this->nnx = static_cast<int>(findClosestProduct(nx, sigma));
        nny = this->nny = static_cast<int>(findClosestProduct(ny, sigma));
        nnz = this->nnz = static_cast<int>(findClosestProduct(nz, sigma));
        Options::nnx = nnx;
        Options::nny = nny;
        Options::nnz = nnz;
    }
    else
    {
        this->nnx = Options::nnx;
        this->nny = Options::nny;
        this->nnz = Options::nnz;
    }
}
void saxsKernel::resetHistogramParameters(std::vector<std::vector<float>> &oc)
{

    auto qcut = Options::Qcut;
    auto dq = Options::Dq;
    int nfx{(nnx % 2 == 0) ? nnx / 2 : nnx / 2 + 1};
    int nfy{(nny % 2 == 0) ? nny / 2 : nny / 2 + 1};
    int nfz{(nnz % 2 == 0) ? nnz / 2 : nnz / 2 + 1};
    float argx{2.0f * (float)M_PI * oc[XX][XX] / sigma};
    float argy{2.0f * (float)M_PI * oc[YY][YY] / sigma};
    float argz{2.0f * (float)M_PI * oc[ZZ][ZZ] / sigma};

    std::vector<float> fx{(float)nfx - 1, (float)nfy - 1, (float)nfz - 1};

    vector<float> mydq0 = {argx, argy, argz, dq};
    vector<float> mycut0 = {argx * fx[XX], argy * fx[YY], argz * fx[ZZ], qcut};

    dq = (*std::max_element(mydq0.begin(), mydq0.end()));
    qcut = *std::min_element(mycut0.begin(), mycut0.end());
    if (qcut != Options::Qcut)
    {
        std::string formatted_string = fmt::format("----- Qcut had to be reset to {:.2f} from  {:.2f} ----", qcut, Options::Qcut);
        std::cout << "\n--------------------------------------------------\n";
        std::cout << formatted_string << "\n";
        std::cout << "--------------------------------------------------\n\n";

        Options::Qcut = qcut;
    }
    if (dq != Options::Dq)
    {
        std::string formatted_string = fmt::format("----- Dq had to be reset to {:.3f} from  {:.3f} ----", dq, Options::Dq);
        std::cout << "\n--------------------------------------------------\n";
        std::cout << formatted_string << "\n";
        std::cout << "--------------------------------------------------\n\n";

        Options::Dq = dq;
    }
}
void saxsKernel::writeBanner()
{
    std::string banner = fmt::format(
        "*************************************************\n"
        "* {:^40}      *\n"
        "* {:<19} {:>4} * {:>4} * {:>4}        *\n"
        "* {:<19} {:>4} * {:>4} * {:>4}        *\n"
        "* {:<10} {:>4}      {:<10} {:>4}          *\n"
        "* {:<10} {:>4.3f}     {:<10} {:>2.f}      *\n"
        "*************************************************\n\n",
        "Running cudaSAXS", "Cell Grid", Options::nx, Options::ny, Options::nz,
        "Supercell Grid", Options::nnx, Options::nny, Options::nnz, "Order",
        Options::order, "Sigma", Options::sigma, "Bin Size", Options::Dq, "Q Cutoff ", Options::Qcut);

    std::cout << banner;
}

saxsKernel::~saxsKernel()
{
}
